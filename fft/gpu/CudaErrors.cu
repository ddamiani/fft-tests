#include "CudaErrors.hh"
#include <iostream>

static const char* cufftGetErrorString(hipfftResult err) {
  switch (err) {
  case HIPFFT_SUCCESS:
    return "HIPFFT_SUCCESS";
  case HIPFFT_INVALID_PLAN:
      return "HIPFFT_INVALID_PLAN";
  case HIPFFT_ALLOC_FAILED:
    return "HIPFFT_ALLOC_FAILED";
  case HIPFFT_INVALID_TYPE:
    return "HIPFFT_INVALID_TYPE";
  case HIPFFT_INVALID_VALUE:
    return "HIPFFT_INVALID_VALUE";
  case HIPFFT_INTERNAL_ERROR:
    return "HIPFFT_INTERNAL_ERROR";
  case HIPFFT_EXEC_FAILED:
    return "HIPFFT_EXEC_FAILED";
  case HIPFFT_SETUP_FAILED:
    return "HIPFFT_SETUP_FAILED";
  case HIPFFT_INVALID_SIZE:
    return "HIPFFT_INVALID_SIZE";
  case HIPFFT_UNALIGNED_DATA:
    return "HIPFFT_UNALIGNED_DATA";
  case HIPFFT_INVALID_DEVICE:
    return "HIPFFT_INVALID_DEVICE";
  case HIPFFT_NO_WORKSPACE:
    return "HIPFFT_NO_WORKSPACE";
  case HIPFFT_NOT_IMPLEMENTED:
    return "HIPFFT_NOT_IMPLEMENTED";
  case HIPFFT_NOT_SUPPORTED:
    return "HIPFFT_NOT_SUPPORTED";
  case CUFFT_MISSING_DEPENDENCY:
    return "CUFFT_MISSING_DEPENDENCY";
  case CUFFT_NVRTC_FAILURE:
    return "CUFFT_NVRTC_FAILURE";
  case CUFFT_NVJITLINK_FAILURE:
    return "CUFFT_NVJITLINK_FAILURE";
  case CUFFT_NVSHMEM_FAILURE:
    return "CUFFT_NVSHMEM_FAILURE";
  default:
    return "Unknown";
  }
}

void fft::gpu::printCudaError(std::string msg, hipError_t err)
{
  std::cerr << " *** " << msg << ": "
            << hipGetErrorString(err)
            << " ***" << std::endl;
}

void fft::gpu::printCudaError(std::string msg, hipfftResult err)
{
  std::cerr << " *** " << msg << ": "
            << cufftGetErrorString(err)
            << " ***" << std::endl;
}
