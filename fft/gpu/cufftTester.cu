#include "hip/hip_runtime.h"
#include "cufftTester.hh"
#include "CudaErrors.hh"
#include "fft/utils/Units.hh"
#include "fft/utils/Signals.hh"
#include <hip/hip_runtime.h>
#include <cmath>
#include <iostream>

using namespace fft::gpu;
using namespace fft::math;

constexpr int MAX_GPUS = 16;

cufftBaseTester::cufftBaseTester(const std::string& name,
                                 unsigned int parallelization,
                                 unsigned int flags,
                                 bool verbose) :
  Tester(name, parallelization, flags, verbose),
  _plan(0),
  _signal(nullptr),
  _result(nullptr),
  _devices()
{
  int ndevices;
  hipError_t err;
  hipDeviceProp_t prop;
  if ((err = hipGetDeviceCount(&ndevices)) != hipSuccess) {
    printCudaError("CUDAError: Problem getting device count", err);
  } else {
    for (int device=0; device<std::min(ndevices, MAX_GPUS); ++device) {
      if ((err = hipGetDeviceProperties(&prop, device)) != hipSuccess) {
        printCudaError("CUDAError: Problem getting device properties", err);
      } else {
        _devices.push_back(device);
        if (this->verbose()) {
          std::cout << " *** GPU Device Properties:" << std::endl
                    << "   Device Number:                " << device << std::endl
                    << "   Device Name:                  " << prop.name << std::endl
                    << "   Compute Capability:           " << prop.major << "." << prop.minor << std::endl
                    << "   Memory Clock Rate (KHz):      " << prop.memoryClockRate << std::endl
                    << "   Memory Bus Width (bits):      " << prop.memoryBusWidth << std::endl
                    << "   Peak Memory Bandwidth (GB/s): "
                    << 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6 << std::endl
                    << "   Total Constant Memory (KB):   " << prop.totalConstMem/1024.0 << std::endl
                    << "   Total Global Memory (GB):     " << prop.totalGlobalMem/pow(1024, 3)
                    << std::endl;
        }
      }
    }
  }
}

cufftBaseTester::~cufftBaseTester()
{
  if (_plan) {
    hipfftDestroy(_plan);
    _plan = 0;
  }
  if (_signal) {
    free(_signal);
    _signal = nullptr;
  }
  if (_result) {
    free(_result);
    _result = nullptr;
  }
}

int cufftBaseTester::ndevices() const
{
  return _devices.size();
}

const int* cufftBaseTester::devices() const
{
  return _devices.data();
}

int* cufftBaseTester::devices()
{
  return _devices.data();
}


bool cufftBaseTester::ready() const
{
  return _plan != 0;
}

bool cufftBaseTester::is_remote() const
{ 
  return true;
}

void cufftBaseTester::destroy_plan()
{
  if (_plan) {
    hipfftDestroy(_plan);
    _plan = 0;
  }
}

void cufftBaseTester::display(unsigned int maxprint) const
{
  if (verbose()) {
    unsigned npoints = min(num_points(), maxprint);

    for (unsigned int i = 0; i < npoints; ++i) {
      std::cout << hipCabs(_result[i]) << std::endl;
    }
  }
}

bool cufftBaseTester::_allocate()
{
  // allocate memory and fill the signal array
  if (cufft_alloc()) {
    unsigned int points_per_batch = num_points();
    unsigned int num_batches = batches();

    for (unsigned int b = 0; b < num_batches; ++b) {
      for (unsigned int i = points_per_batch * b; i < points_per_batch * (b + 1); ++i) {
        signal(_signal[i].x, _signal[i].y, (double)i / (double)points_per_batch);
      }
    }
    return true;
  } else {
    return false;
  }
}

bool cufftBaseTester::cufft_alloc()
{
  if (cufft_device_alloc()) {
    return cufft_host_alloc();
  } else {
    return false;
  }
}

bool cufftBaseTester::cufft_host_alloc()
{
  size_t size = num_points() * sizeof(hipfftDoubleComplex) * batches();
  // free the signal and result if they already exist
  if (_signal) free(_signal);
  if (_result) free(_result);

  // allocate memory on the host
  _signal = reinterpret_cast<hipfftDoubleComplex*>(malloc(size));
  _result = reinterpret_cast<hipfftDoubleComplex*>(malloc(size));

  return _signal && _result;
}

cufftTester::cufftTester(const std::string& name,
                         unsigned int flags,
                         bool verbose) :
  cufftBaseTester(name, 1, flags, verbose),
  _dev_data(nullptr)
{}

cufftTester::~cufftTester()
{
  if (_dev_data) {
    hipFree(_dev_data);
    _dev_data = nullptr;
  }
}

bool cufftTester::send_data() {
  size_t size = num_points() * sizeof(hipfftDoubleComplex);
  hipError_t err = hipMemcpy(_dev_data, _signal, size, hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    printCudaError("CUDAError: Problem copying data to device", err);
    return false;
  } else {
    return true;
  }
}

bool cufftTester::execute()
{
  hipfftResult ffterr = hipfftExecZ2Z(_plan, _dev_data, _dev_data, HIPFFT_FORWARD);
  if (ffterr != HIPFFT_SUCCESS) {
    printCudaError("cuFFT Error: Problem executing the plan", ffterr);
    return false;
  }

  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printCudaError("CUDAError: Problem synchronizing device", err);
    return false;
  } else {
    return true;
  }
}

bool cufftTester::recv_data()
{
  size_t size = num_points() * sizeof(hipfftDoubleComplex);
  hipError_t err = hipMemcpy(_result, _dev_data, size, hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    printCudaError("CUDAError: Problem copying data to host", err);
    return false;
  } else {
    return true;
  }
}

bool cufftTester::_create_plan()
{
  size_t workSize = 0;
  unsigned int npoints = num_points();

  // create the plan
  hipfftResult err = hipfftPlanMany(&_plan, rank(), shape(),
                                  NULL, 1, npoints,
                                  NULL, 1, npoints,
                                  HIPFFT_Z2Z, batches());
  if (err != HIPFFT_SUCCESS) {
    printCudaError("cuFFT Error: Unable to create plan", err);
    return false;
  } else if ((err = hipfftGetSize(_plan, &workSize)) != HIPFFT_SUCCESS) {
    printCudaError("cuFFT Error: Unable to get plan work size", err);
    return false;
  } else {
    if (verbose()) {
      Value<double> workSizeConv = convert_bytes<double>(workSize);
      std::cout << " *** Created plan with work area size (" << workSizeConv.unit
                << "): " <<  workSizeConv.value << " ***" << std::endl;
    }
    return true;
  }
}

bool cufftTester::cufft_device_alloc()
{
  hipError_t err;
  size_t size = num_points() * sizeof(hipfftDoubleComplex) * batches();

  // free the device memory if needed
  if (_dev_data) {
    if ((err = hipFree(_dev_data)) != hipSuccess) {
      printCudaError("CUDAError: failure freeing device memory", err);
      return false;
    } else {
      _dev_data = nullptr;
    }
  }

  // allocate memory on the device
  if ((err = hipMalloc((void**)&_dev_data, size)) != hipSuccess) {
    printCudaError("CUDAError: failure allocating device memory", err);
    return false;
  } else {
    return true;
  }
}

cufftXtTester::cufftXtTester(const std::string& name,
                             unsigned int parallelization,
                             unsigned int flags,
                             bool verbose) :
  cufftBaseTester(name, parallelization, flags, verbose),
  _dev_data(nullptr)
{}

cufftXtTester::~cufftXtTester()
{
  if (_dev_data) {
    hipfftXtFree(_dev_data);
    _dev_data = nullptr;
  }
}

bool cufftXtTester::send_data() {
  size_t size = num_points() * sizeof(hipfftDoubleComplex);
  hipfftResult err = hipfftXtMemcpy(_plan, _dev_data, _signal, HIPFFT_COPY_HOST_TO_DEVICE);
  if (err != HIPFFT_SUCCESS) {
    printCudaError("cuFFT Error: Problem copying data to device", err);
    return false;
  } else {
    return true;
  }
}

bool cufftXtTester::execute()
{
  hipfftResult ffterr = hipfftXtExecDescriptorZ2Z(_plan, _dev_data, _dev_data, HIPFFT_FORWARD);
  if (ffterr != HIPFFT_SUCCESS) {
    printCudaError("cuFFT Error: Problem executing the plan", ffterr);
    return false;
  }

  hipError_t err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printCudaError("CUDAError: Problem synchronizing device", err);
    return false;
  } else {
    return true;
  }
}

bool cufftXtTester::recv_data()
{
  size_t size = num_points() * sizeof(hipfftDoubleComplex);
  hipfftResult err = hipfftXtMemcpy(_plan, _result, _dev_data, HIPFFT_COPY_HOST_TO_DEVICE);
  if (err != HIPFFT_SUCCESS) {
    printCudaError("cuFFT Error: Problem copying data to host", err);
    return false;
  } else {
    return true;
  }
}

bool cufftXtTester::_set_num_gpu()
{
  int requested_dev = parallelization();

  // check if enough gpus are available
  if (requested_dev > ndevices()) {
    std::cerr << " *** Requested number of GPUs (" << requested_dev
              << ") is higher than number available ("  << ndevices()
              << ")! ***" << std::endl;
    return false;
  }

  hipfftResult err = hipfftXtSetGPUs(_plan, requested_dev, devices());
  if (err != HIPFFT_SUCCESS) {
    printCudaError("cuFFT Error: Unable to set the number of GPUs", err);
    return false;
  } else {
    return true;
  }
}

bool cufftXtTester::_alloc_needs_plan() const
{
  return true;
}

bool cufftXtTester::_create_plan()
{
  size_t workSize = 0;
  unsigned int npoints = num_points();

  // create and empty plan
  hipfftResult err = hipfftCreate(&_plan);
  if (err != HIPFFT_SUCCESS) {
    printCudaError("cuFFT Error: Unable to create empty plan", err);
    return false;
  } else if (!_set_num_gpu()) {
    std::cerr << " *** Failed to set the "
              << "number of GPUs used for the FFT to "
              << parallelization() << "! ***" << std::endl;
    return false;
  } else {
    // create the plan
    hipfftResult err = hipfftMakePlanMany(_plan, rank(), shape(),
                                        NULL, 1, npoints,
                                        NULL, 1, npoints,
                                        HIPFFT_Z2Z, batches(), &workSize);
    if (err != HIPFFT_SUCCESS) {
      printCudaError("cuFFT Error: Unable to create plan", err);
      return false;
    } else {
      if (verbose()) {
        Value<double> workSizeConv = convert_bytes<double>(workSize);
        std::cout << " *** Created plan with work area size (" << workSizeConv.unit
                  << "): " <<  workSizeConv.value << " ***" << std::endl;
      }
      return true;
    }
  }
}

bool cufftXtTester::cufft_device_alloc()
{
  hipfftResult err;
  size_t size = num_points() * sizeof(hipfftDoubleComplex) * batches();

  // free the device memory if needed
  if (_dev_data) {
    if ((err = hipfftXtFree(_dev_data)) != HIPFFT_SUCCESS) {
      printCudaError("cuFFT Error: failure freeing device memory", err);
      return false;
    } else {
      _dev_data = nullptr;
    }
  }

  // allocate memory on the device
  if ((err = hipfftXtMalloc(_plan, &_dev_data, HIPFFT_XT_FORMAT_INPLACE)) != HIPFFT_SUCCESS) {
    printCudaError("cuFFT Error: failure allocating device memory", err);
    return false;
  } else {
    return true;
  }
}
